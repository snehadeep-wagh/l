#include <bits/stdc++.h>
#include <hip/hip_runtime.h>
#define BLOCK_SIZE 64

using namespace std;

void cpu_add(int *arr1, int *arr2, int *res, int size)
{
    for (int i = 0; i < size; ++i)
        res[i] = arr1[i] + arr2[i];
}

__global__ void gpu_add(int *arr1, int *arr2, int *res, int size)
{
    int block_id = blockIdx.x * blockDim.x + threadIdx.x;
    if (block_id < size)
    {
        res[block_id] = arr1[block_id] + arr2[block_id];
    }
}

void print_vec(int *arr, int size)
{
    for (int i = 0; i < size; ++i)
        cout << arr[i] << " ";
    cout << endl;
}

int main()
{
    int *a_cpu, *b_cpu, *res_cpu, *a_gpu, *b_gpu, *res_gpu, *copy;
    int n = 1 << 4; // 16

    // new memory is allocated to every array
    a_cpu = new int[n];
    b_cpu = new int[n];
    res_cpu = new int[n];
    copy = new int[n];
    int size = n * sizeof(int);
    hipMalloc(&a_gpu, size);
    hipMalloc(&b_gpu, size);
    hipMalloc(&res_gpu, size);

    // generate array randomly
    for (int i = 0; i < n; ++i)
    {
        a_cpu[i] = rand() % 100;
        b_cpu[i] = rand() % 100;
    }

    // transfer data from the host (CPU) to the device (GPU). 
    hipMemcpy(a_gpu, a_cpu, size, hipMemcpyHostToDevice);
    hipMemcpy(b_gpu, b_cpu, size, hipMemcpyHostToDevice);

    cout << "\nVector A => ";
    print_vec(a_cpu, n);
    cout << "\nVector B => ";
    print_vec(b_cpu, n);

    hipEvent_t start, stop;
    float elapsedTime;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    // records the occurrence of the specified event in the specified stream. 
    //By recording an event, 
    //you can later use it to measure the time elapsed between events or 
    //to synchronize different parts of your CUDA program.
    // 0 ->  default execution stream in CUDA.
    hipEventRecord(start, 0);
    cpu_add(a_cpu, b_cpu, res_cpu, n);
    hipEventRecord(stop, 0);

    /*The cudaEventSynchronize function ensures that the CPU (host) waits 
    until the specified CUDA event is complete. It provides synchronization 
    between the CPU and GPU, ensuring that the CPU does not proceed further 
    until the GPU has finished all preceding operations recorded 
    before the specified event.*/
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    cout << "\nCPU result:" << endl;
    print_vec(res_cpu, n);
    cout << "Elapsed Time = " << elapsedTime << " ms" << endl;

    int threads = 1024;
    int blocks = (n + threads - 1) / threads;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    gpu_add<<<blocks, threads>>>(a_gpu, b_gpu, res_gpu, n);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipMemcpy(copy, res_gpu, size, hipMemcpyDeviceToHost);
    cout << "\nGPU result:" << endl;
    print_vec(copy, n);
    cout << "Elapsed Time = " << elapsedTime << " ms" << endl;

    hipFree(a_gpu);
    hipFree(b_gpu);
    hipFree(res_gpu);

    return 0;
}